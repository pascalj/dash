
#include "ForEachTest.h"

#include <dash/Array.h>
#include <dash/Matrix.h>
#include <dash/SharedCounter.h>
#include <dash/algorithm/Fill.h>
#include <dash/algorithm/ForEach.h>
#include <dash/algorithm/Reduce.h>

#include <dash/Mephisto.h>
#include <patterns/local_pattern.h>

#include <functional>


TEST_F(ForEachTest, TestArrayAllInvoked) {
    // Shared variable for total number of invoked callbacks:
    dash::SharedCounter<size_t> count_invokes;
    // Create for_each callback from member function:
    std::function<void(const Array_t::value_type &)> invoke =
        std::bind(&ForEachTest::count_invoke, this, std::placeholders::_1);
    // Ensure value global counter is published to all units
    dash::Team::All().barrier();
    // Initialize global array:
    Array_t array(_num_elem);
    // Run for_each on complete array
    dash::for_each(array.begin(), array.end(), invoke);
    // Should have been invoked on every local index in the array:
    LOG_MESSAGE("Local number of inspected indices: %zu",
                _invoked_indices.size());
    EXPECT_EQ(array.lsize(), _invoked_indices.size());
    // Count number of local invokes
    count_invokes.inc(_invoked_indices.size());
    // Wait for all units
    array.barrier();
    // Test number of total invokes
    size_t num_invoked_indices_all = count_invokes.get();
    LOG_MESSAGE("Total number of inspected indices: %zu",
                num_invoked_indices_all);
    EXPECT_EQ(_num_elem, num_invoked_indices_all);
}

TEST_F(ForEachTest, ForEachWithIndex)
{
    if (dash::size() == 3) {
      // TODO: Fix this
      SKIP_TEST();
    }

    std::function<void(const Element_t &, index_t)> dummy_fct =
    [](Element_t el, index_t idx) {
        EXPECT_EQ_U(
            el,
            static_cast<Element_t>(dash::myid())
        );
    };

    // Test Array
    Array_t array(_num_elem);
    dash::fill(
        array.begin(),
        array.end(),
        static_cast<Element_t>(dash::myid()));

    dash::for_each_with_index(
        array.begin(),
        array.end(),
        dummy_fct);

    // Test Matrix
    dash::Matrix<Element_t, 2> matrix(dash::SizeSpec<2>(
                                        dash::size(),
                                        dash::size()));
    dash::fill(
        matrix.begin(),
        matrix.end(),
        static_cast<Element_t>(dash::myid()));

    dash::for_each_with_index(
        matrix.begin(),
        matrix.end(),
        dummy_fct);
}

TEST_F(ForEachTest, ForEachWithIndexPos)
{
  dash::Array<int> array(100, dash::CYCLIC);

  // Fill
  std::function< void(const int &, index_t)>
  fill = [&array](int el, index_t i) {
    auto coords = array.pattern().coords(i);
    array[i] = coords[0];
  };

  // Verify
  std::function< void(const int &, index_t)>
    verify = [&array](int el, index_t i) {
      auto coords  = array.pattern().coords(i);
      auto desired = coords[0];
      ASSERT_EQ_U(
        desired,
        el);
    };

  // Fill
  dash::for_each_with_index(
    array.begin(),
    array.end(),
    fill);

  dash::for_each_with_index(
    array.begin(),
    array.end(),
    verify);
}

TEST_F(ForEachTest, ModifyValues)
{
  dash::Array<int> array(100, dash::TILE(10));
  dash::fill(array.begin(), array.end(), dash::myid());

  std::function< void(int &)>
    incr = [](int & el) {
      el = el+1;
  };
  std::function< void(const int & )>
    verify = [](const int & el){
      ASSERT_EQ_U(el, dash::myid()+1);
  };

  // Increment by one
  dash::for_each(array.begin(), array.end(), incr);
  // Verify
  dash::for_each(array.begin(), array.end(), verify);
}

TEST_F(ForEachTest, Lambdas)
{
  dash::Array<int> array(100, dash::TILE(10));
  dash::fill(array.begin(), array.end(), dash::myid());

  // -- dash::for_each -------------------------------------------

  // Increment by 100:
  dash::for_each(array.begin(), array.end(),
                 [](int & el) {
                   el += 100;
                 });
  // Verify:
  dash::for_each(array.begin(), array.end(),
                 [](int & el) {
                   EXPECT_EQ_U(100 + dash::myid(), el);
                 });

  // -- dash::for_each_with_index --------------------------------

  // Increment by element index:
  dash::for_each_with_index(
                 array.begin(), array.end(),
                 [](int & el, index_t gindex) {
                   el += gindex;
                 });
  // Verify:
  dash::for_each_with_index(
                 array.begin(), array.end(),
                 [](int & el, index_t gindex) {
                   EXPECT_EQ_U(100 + dash::myid() + gindex, el);
                 });
}

#ifdef ALPAKA_ACC_CPU_B_SEQ_T_SEQ_ENABLED
TEST_F(ForEachTest, MephistoBasicTest)
{
  using value_t   = int;
  using entity_t  = dash::CpuThreadEntity<1>;
  using pattern_t =
      patterns::BalancedLocalPattern<dash::BlockPattern<1>, entity_t>;

  pattern_t pattern{100 * dash::size()};
  const auto layout = dash::ROW_MAJOR;
  dash::NArray<value_t, 1, pattern_t::index_type, pattern_t> arr(pattern);

  dash::AlpakaExecutor<entity_t> executor;

  dash::fill(arr.begin(), arr.end(), 51);

  dash::for_each(
      executor, arr.begin(), arr.end(), [](int& a) {
        return a = a * 111;
      });

  auto sum = dash::reduce(arr.begin(), arr.end(), 0);
  EXPECT_EQ_U(dash::size() * 51 * 100 * 111, sum);
}
#endif

#ifdef ALPAKA_ACC_GPU_CUDA_ENABLED

void run_my_test() {
  using value_t   = uint64_t;
  using entity_t  = dash::CudaEntity<1>;
  using pattern_t =
      patterns::UnitRRPattern<dash::BlockPattern<1>, entity_t>;

  using memory_t = dash::CudaSpace;

  const size_t problem_size = 64 * 1024 * 1024 * dash::size() / sizeof(value_t);
  const value_t init = 51;

  pattern_t pattern{problem_size};
  dash::NArray<value_t, 1, pattern_t::index_type, pattern_t, memory_t> arr(pattern);

  dash::AlpakaExecutor<entity_t> executor;

  dash::fill(arr.begin(), arr.end(), init);

#ifdef __CUDACC_EXTENDED_LAMBDA__
  auto times111 = [=] __device__ (value_t &a) { return a *= 111; };
#else
#error "Need extended lambdas."
#endif
  dash::for_each(
      executor, arr.begin(), arr.end(), times111);

  hipDeviceSynchronize();
  long long sum = dash::reduce(arr.begin(), arr.end(), 0LL);
  EXPECT_EQ(problem_size * init * 111, sum);
}


TEST_F(ForEachTest, MephistoCudaBasicTest)
{
  run_my_test();
}
#endif
